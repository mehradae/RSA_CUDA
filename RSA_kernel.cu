
#include <hip/hip_runtime.h>
__device__ long long int mod(int base, int exponent, int den) {

	long long int ret;
	ret = 1;
	for (int i = 0; i < exponent; i++) {
		ret *= base;
		ret = ret % den;
	}
	return ret;

}

__device__ long long int mod_optimized(int base, int exponent, int den) {

	unsigned int a = (base % den) * (base % den);
	unsigned long long int ret = 1;
	float size = (float) exponent / 2;
	if (exponent == 0) {
		return base % den;
	} else {
		while (1) {
			if (size > 0.5) {
				ret = (ret * a) % den;
				size = size - 1.0;
			} else if (size == 0.5) {
				ret = (ret * (base % den)) % den;
				break;
			} else {
				break;
			}
		}
		return ret;
	}

}

__global__ void rsa(int * num, int *key, int *den, int * result) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int temp;
	//temp = mod(num[i], *key, *den);
	temp = mod_optimized(num[i], *key, *den);
	atomicExch(&result[i], temp);
}
