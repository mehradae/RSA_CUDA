#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include "RSA_kernel.cu"
#define BUZZ_SIZE 10002

int p, q, n, t, flag, e[100], d[100], temp[BUZZ_SIZE], j, m[BUZZ_SIZE],
		en[BUZZ_SIZE], mm[BUZZ_SIZE], res[BUZZ_SIZE], i;
char msg[BUZZ_SIZE];
int prime(long int);
void generate_input(int);
void ce();
long int cd(long int);
void encrypt();
void decrypt();
void encrypt_gpu();
void decrypt_gpu();
int numChars;
int input;
int threadsPerBlock = 1024;
int blocksPerGrid;
time_t tt;
double time_encrypt_cpu, time_decrypt_cpu;
float time_encrypt_gpu = 0.0;
float time_decrypt_gpu = 0.0;

int main() {

	p = 157;
	q = 373;



	printf("\n\n\n\n******************************************************************\n");
	printf("*****  Welcome to  Encryption/Decryption program Using RSA  ******\n");
	printf("******************************************************************\n\n");

	printf("Please select one of these options:\n");
	printf("1. Enter your own input\n2.Random generated input\n\n");

	input=fgetc(stdin);
	input=input-'0';
	// Check if the input is valid
	if(input!=1 && input !=2)
	{
		printf("This is a wrong amount of input!\nQuiting now ...\n");
		exit(0);
	}
	if(input==1)
	{

	 printf("\nENTER MESSAGE: ");
	 scanf("%s",msg);
	 numChars = strlen(msg);
	 blocksPerGrid =(numChars + threadsPerBlock - 1) / threadsPerBlock;
	
	}
	else
	{
	srand((unsigned) time(&tt));/* Intializes random number generator */
	generate_input(10000);
	
	FILE *f = fopen("input.txt", "r");
	if (f == NULL) {
		perror("Error opening file");
		return (1);
	}
	if (fgets(msg, BUZZ_SIZE, f) != NULL) {
		//printf("String read: %s\n", msg);
		printf("Reading input file...done");
	}
	fclose(f);

	numChars = strlen(msg) - 1;
	printf("numChars: %d)\n\n", numChars);
	blocksPerGrid = (numChars + threadsPerBlock - 1) / threadsPerBlock;
	}

	for (i = 0; i<numChars; i++) {
		m[i] = msg[i];
		mm[i] = msg[i] - 96;
	}
	n = p * q;
	t = (p - 1) * (q - 1);
	ce();
	/*
	 printf("\nPOSSIBLE VALUES OF e AND d ARE\n");
	 for (i = 0; i < j - 1; i++)
	 printf("\n%ld\t%ld", e[i], d[i]);
	 */

	encrypt();
	decrypt();
	encrypt_gpu();
	decrypt_gpu();
	printf("GPU encryption speed up: %f times\n",
			time_encrypt_cpu / time_encrypt_gpu);
	printf("GPU decryption speed up: %f times\n\n",
			time_decrypt_cpu / time_decrypt_gpu);

	
	return 0;
}

void generate_input(int size) {
	printf("\nGenerating input file...\n ");
	FILE *fp = fopen("input.txt", "wb");
	if (fp != NULL) {
		for (int k = 0; k < size; k++) {
			int r = rand() % 26;
			fprintf(fp, "%c", r + 97);
		}
		fprintf(fp, "\n");
		fclose(fp);
		printf("done\n");
	}
}

int prime(long int pr) {
	int i;
	j = sqrt(pr);
	for (i = 2; i <= j; i++) {
		if (pr % i == 0)
			return 0;
	}
	return 1;
}

void ce() {
	int k;
	k = 0;
	for (i = 2; i < t; i++) {
		if (t % i == 0)
			continue;
		flag = prime(i);
		if (flag == 1 && i != p && i != q) {
			e[k] = i;
			long int z = 1;
			while (1) {
				z = z + t;
				if (z % e[k] == 0){
					flag = z / e[k];
					break;
				}
			}
			if (flag > 0) {
				d[k] = flag;
				k++;
			}
			if (k == 99)
				break;
		}
	}
}


void encrypt() {
	double start_encrypt, end_encrypt;
	start_encrypt = clock();
	printf("CPU starts encrypting...\n");
	int pt, ct, key = e[0], k, len;
	printf("\ne=%d\n",key);
	i = 0;
	len = numChars;
	while (i != len) {
		pt = m[i];
		pt = pt - 96;
		k = 1;
		for (j = 0; j < key; j++) {
			k = k * pt;
			k = k % n;
		}
		temp[i] = k;
		ct = k + 96;
		en[i] = ct;
		i++;
	}
	end_encrypt = clock();
	time_encrypt_cpu = (double) (end_encrypt - start_encrypt) / CLOCKS_PER_SEC;
	printf("Encryption time taken by CPU: %f s\n", time_encrypt_cpu);
	/*
	 en[i] = -1;
	 printf("\nCPU ENCRYPTED MESSAGE IS\n");
	 for (i = 0; en[i] != -1; i++)
	 printf("%d ", en[i]);
	 */

	printf("Saving CPU encrypted file in encrypted_cpu.txt\n ");
	en[i] = -1;
	FILE *fp = fopen("encrypted_cpu.txt", "wb");
	if (fp != NULL) {
		for (int k = 0; en[k] != -1; k++) {
			fprintf(fp, "%d", en[k]);
		}
		fclose(fp);
		printf("done\n\n");
	}
}

void encrypt_gpu() {
	hipEvent_t start_encrypt, stop_encrypt;
	int key = e[0];
	//printf("\nkey=%d, n=%d\n",key,n);
	hipSetDevice(1);
	int *dev_num, *dev_key, *dev_den;
	int *dev_res;
	hipMalloc((void **) &dev_num, numChars * sizeof(int));
	hipMalloc((void **) &dev_key, sizeof(int));
	hipMalloc((void **) &dev_den, sizeof(int));
	hipMalloc((void **) &dev_res, numChars * sizeof(int));
	hipMemcpy(dev_num, mm, numChars * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_key, &key, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_den, &n, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_res, res, numChars * sizeof(int), hipMemcpyHostToDevice);

	hipEventCreate(&start_encrypt);
	hipEventCreate(&stop_encrypt);
	hipEventRecord(start_encrypt);
	printf("GPU starts encrypting...\n");
	rsa<<<blocksPerGrid, threadsPerBlock>>>(dev_num,dev_key,dev_den,dev_res);
	hipEventRecord(stop_encrypt);
	hipEventSynchronize(stop_encrypt);
	hipDeviceSynchronize();
	hipEventElapsedTime(&time_encrypt_gpu, start_encrypt, stop_encrypt);

	hipMemcpy(res, dev_res, numChars * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_num);
	hipFree(dev_key);
	hipFree(dev_den);
	hipFree(dev_res);

	time_encrypt_gpu /= 1000;
	printf("Encryption time taken by GPU: %f s\n", time_encrypt_gpu);

	/*
	 printf("\nGPU ENCRYPTED MESSAGE IS\n");
	 for (i = 0; i < numChars; i++)
	 printf("%d ", res[i]+96);
	 printf("\n");
	 */

	printf("Saving GPU encrypted file in encrypted_gpu.txt\n");
	FILE *fp = fopen("encrypted_gpu.txt", "wb");
	if (fp != NULL) {
		for (i = 0; i < numChars; i++) {
			fprintf(fp, "%d", res[i] + 96);
		}
		fclose(fp);
		printf("done\n\n");
	}
}

void decrypt_gpu() {
	hipEvent_t start_decrypt, stop_decrypt;
	int key = d[0];
	//printf("\nkey=%d, n=%d\n",key,n);
	hipSetDevice(1);
	int *dev_num, *dev_key, *dev_den;
	int *dev_res;
	hipMalloc((void **) &dev_num, numChars * sizeof(int));
	hipMalloc((void **) &dev_key, sizeof(int));
	hipMalloc((void **) &dev_den, sizeof(int));
	hipMalloc((void **) &dev_res, numChars * sizeof(int));
	hipMemcpy(dev_num, res, numChars * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_key, &key, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_den, &n, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_res, res, numChars * sizeof(int), hipMemcpyHostToDevice);

	hipEventCreate(&start_decrypt);
	hipEventCreate(&stop_decrypt);
	hipEventRecord(start_decrypt);
	printf("GPU starts decrypting...\n");
	rsa<<<blocksPerGrid, threadsPerBlock>>>(dev_num,dev_key,dev_den,dev_res);
	hipEventRecord(stop_decrypt);
	hipEventSynchronize(stop_decrypt);
	hipDeviceSynchronize();
	hipEventElapsedTime(&time_decrypt_gpu, start_decrypt, stop_decrypt);

	hipMemcpy(res, dev_res, numChars * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_num);
	hipFree(dev_key);
	hipFree(dev_den);
	hipFree(dev_res);

	time_decrypt_gpu /= 1000;
	printf("Decryption time taken by GPU: %f s\n", time_decrypt_gpu);

	/*
	 printf("\nGPU DECRYPTED MESSAGE IS\n");
	 for (i = 0; i < numChars; i++)
	 printf("%d ", res[i]+96);
	 printf("\n");
	 */

	printf("Saving GPU decrypted file in decrypted_gpu.txt\n");
	FILE *fp = fopen("decrypted_gpu.txt", "wb");
	if (fp != NULL) {
		for (i = 0; i < numChars; i++) {
			fprintf(fp, "%c", res[i] + 96);
		}
		fprintf(fp, "\n");
		fclose(fp);
		printf("done\n\n");
	}
}

void decrypt() {
	double start_decrypt, end_decrypt;
	start_decrypt = clock();
	printf("CPU starts decrypting...\n");
	long int pt, ct, key = d[0], k;
	printf("\nd=%d\n",key);
	i = 0;
	while (en[i] != -1) {
		ct = temp[i];
		k = 1;
		for (j = 0; j < key; j++) {
			k = k * ct;
			k = k % n;
		}
		pt = k + 96;
		m[i] = pt;
		i++;
	}
	end_decrypt = clock();
	time_decrypt_cpu = (double) (end_decrypt - start_decrypt) / CLOCKS_PER_SEC;
	printf("Decryption time taken by CPU: %f s\n", time_decrypt_cpu);

	/*
	 m[i] = -1;
	 printf("\nCPU DECRYPTED MESSAGE IS\n");
	 for (i = 0; m[i] != -1; i++)
	 printf("%d ", m[i]);
	 printf("\n");
	 */

	printf("Saving CPU decrypted file in decrypted_cpu.txt\n");
	m[i] = -1;
	FILE *fp = fopen("decrypted_cpu.txt", "wb");
	if (fp != NULL) {
		for (int k = 0; m[k] != -1; k++) {
			fprintf(fp, "%c", m[k]);
		}
		fprintf(fp, "\n");
		fclose(fp);
		printf("done\n\n");
	}
}
